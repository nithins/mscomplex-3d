#include "hip/hip_runtime.h"
#include <cutil.h>

texture<flag_t, 3, hipReadModeElementType>     flag_texture;
texture<func_t, 3, hipReadModeElementType>     func_texture;

__device__ bool compare_verts(cell_t v1, cell_t v2)
{
  func_t f1 = tex3D(func_texture, v1.x/2, v1.y/2, v1.z/2);
  func_t f2 = tex3D(func_texture, v2.x/2, v2.y/2, v2.z/2);

  if( f1 != f2)
    return f1 < f2;

  return v1 < v2;
}

__device__ bool compare_edges(cell_t e1, cell_t e2)
{
  flag_t f1 = tex3D(flag_texture, e1.x, e1.y, e1.z);
  flag_t f2 = tex3D(flag_texture, e2.x, e2.y, e2.z);

  cell_t v1 = flag_to_mxfct(e1,f1);
  cell_t v2 = flag_to_mxfct(e2,f2);

  if( v1 == v2)
  {
    v1 = second_max_facet(e1,v1);
    v2 = second_max_facet(e2,v2);
  }

  return compare_verts(v1,v2);
}

__device__ bool compare_faces(cell_t fc1, cell_t fc2)
{
  flag_t f1 = tex3D(flag_texture, fc1.x, fc1.y, fc1.z);
  flag_t f2 = tex3D(flag_texture, fc2.x, fc2.y, fc2.z);

  cell_t e1 = flag_to_mxfct(fc1,f1);
  cell_t e2 = flag_to_mxfct(fc1,f2);

  if( e1 == e2)
  {
    e1 = second_max_facet(fc1,e1);
    e2 = second_max_facet(fc2,e2);
  }

  return compare_edges(e1,e2);
}

__device__ bool compare_cubes(cell_t c1, cell_t c2)
{
  flag_t f1 = tex3D(flag_texture, c1.x, c1.y, c1.z);
  flag_t f2 = tex3D(flag_texture, c2.x, c2.y, c2.z);

  cell_t fc1 = flag_to_mxfct(c1,f1);
  cell_t fc2 = flag_to_mxfct(c1,f2);

  if( fc1 == fc2)
  {
    fc1 = second_max_facet(c1,fc1);
    fc2 = second_max_facet(c2,fc2);
  }

  return compare_faces(fc1,fc2);
}

__global__  void assign_maxfacet_vert
    (const rect_t ext_rect,
     flag_t *cell_flags)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int num_thds= gridDim.x*blockDim.x;

  rect_t cell_rect(ext_rect.lc,ext_rect.uc);

  for( int i = tid ; i < cell_rect.num_cells2(); i += num_thds)
  {
    cell_t c = cell_rect.i_to_c2(i);

    cell_flags[ext_rect.c_to_i(c)] = 0;
  }
}


__global__  void assign_maxfacet_edge
    (const rect_t ext_rect,
     const cell_t edir,
     flag_t *cell_flags)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int num_thds= gridDim.x*blockDim.x;

  rect_t cell_rect(ext_rect.lc+edir,ext_rect.uc-edir);

  for( int i = tid ; i < cell_rect.num_cells2(); i += num_thds)
  {
    cell_t c = cell_rect.i_to_c2(i);

    cell_t v1 = c - edir;
    cell_t v2 = c + edir;

    cell_t v = v1;

    if( compare_verts(v,v2))
      v = v2;

    cell_flags[ext_rect.c_to_i(c)] = mxfct_to_flag(c,v);
  }
}

__global__ void assign_maxfacet_face
    (const rect_t ext_rect,
     const cell_t fdir1,
     const cell_t fdir2,
     flag_t *cell_flags)

{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int num_thds= gridDim.x*blockDim.x;

  cell_t fdir = fdir1+fdir2;
  rect_t cell_rect(ext_rect.lc+fdir,ext_rect.uc-fdir);

  for( int i = tid ; i < cell_rect.num_cells2(); i += num_thds)
  {
    cell_t c = cell_rect.i_to_c2(i);

    cell_t e1 = c-fdir1;
    cell_t e2 = c+fdir1;
    cell_t e3 = c-fdir2;
    cell_t e4 = c+fdir2;

    cell_t e = e1;

    if( compare_edges(e,e2))
      e = e2;

    if( compare_edges(e,e3))
      e = e3;

    if( compare_edges(e,e4))
      e = e4;

    cell_flags[ext_rect.c_to_i(c)] = mxfct_to_flag(c,e);
  }
}

__global__  void assign_maxfacet_cube
    (const rect_t ext_rect,
     flag_t *cell_flags)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int num_thds= gridDim.x*blockDim.x;

  rect_t cell_rect(ext_rect.lc+1,ext_rect.uc-1);

  for( int i = tid ; i < cell_rect.num_cells2(); i += num_thds)
  {
    cell_t c = cell_rect.i_to_c2(i);

    cell_t f1 = c-mk_cell(1,0,0);
    cell_t f2 = c+mk_cell(1,0,0);
    cell_t f3 = c-mk_cell(0,1,0);
    cell_t f4 = c+mk_cell(0,1,0);
    cell_t f5 = c-mk_cell(0,0,1);
    cell_t f6 = c+mk_cell(0,0,1);

    cell_t f = f1;

    if( compare_faces(f,f2))
      f = f2;

    if( compare_faces(f,f3))
      f = f3;

    if( compare_faces(f,f4))
      f = f4;

    if( compare_faces(f,f5))
      f = f5;

    if( compare_faces(f,f6))
      f = f6;

    cell_flags[ext_rect.c_to_i(c)] = mxfct_to_flag(c,f);
  }
}


extern "C"
void assign_max_facet_cuda(short* erptr,func_t *func , flag_t *flag)
{
  rect_t ext_rect(erptr[0],erptr[1],erptr[2],erptr[3],erptr[4],erptr[5]);

  hipArray *d_func = 0;
  cell_t func_span = (ext_rect.uc - ext_rect.lc)/2 + 1;

  hipExtent func_vol_extent = make_hipExtent(func_span.x * sizeof(func_t),func_span.y,func_span.z);

  // create 3D array
  hipChannelFormatDesc func_channel_desc = hipCreateChannelDesc<func_t>();
  cutilSafeCall(hipMalloc3DArray(&d_func, &func_channel_desc, func_vol_extent));

  // pitched pointer
  hipPitchedPtr func_pitched_ptr;
  func_pitched_ptr.ptr      = func;
  func_pitched_ptr.pitch    = func_vol_extent.width*sizeof(func_t);
  func_pitched_ptr.xsize    = func_vol_extent.width;
  func_pitched_ptr.ysize    = func_vol_extent.height;

  // copy data to 3D array
  hipMemcpy3DParms func_copy_params = {0};
  func_copy_params.srcPtr   = func_pitched_ptr;
  func_copy_params.dstArray = d_func;
  func_copy_params.extent   = func_vol_extent;
  func_copy_params.kind     = hipMemcpyHostToDevice;
  cutilSafeCall(hipMemcpy3D(&func_copy_params));

  // set texture parameters
  func_texture.normalized = false;                      // access with normalized texture coordinates
  func_texture.filterMode = hipFilterModePoint;      // linear interpolation
  func_texture.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
  func_texture.addressMode[1] = hipAddressModeClamp;

  // bind array to 3D texture
  cutilSafeCall(hipBindTextureToArray(func_texture, d_func, func_channel_desc));



  flag_t *d_flag = 0;
  int     d_flag_size = ext_rect.num_cells()*sizeof(flag_t);

  cutilSafeCall(hipMalloc(&d_flag, d_flag_size));


  flag_vol_extent.width  = flag_span.x;
  flag_vol_extent.height = flag_span.y;
  flag_vol_extent.depth  = flag_span.z;

  // create 3D array
  hipChannelFormatDesc flag_channel_desc = hipCreateChannelDesc<flag_t>();


  // set texture parameters
  flag_texture.normalized = false;                      // access with normalized texture coordinates
  flag_texture.filterMode = hipFilterModePoint;      // linear interpolation
  flag_texture.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
  flag_texture.addressMode[1] = hipAddressModeClamp;

  // bind array to 3D texture
  cutilSafeCall(hipBindTexture(flag_texture, d_flag, flag_channel_desc));



  dim3 block(256,1);
  dim3 grid(32,1);

  assign_maxfacet_vert<<<grid,block>>>(ext_rect,d_flag);

  assign_maxfacet_edge<<<grid,block>>>(ext_rect,mk_cell(1,0,0),d_flag);
  assign_maxfacet_edge<<<grid,block>>>(ext_rect,mk_cell(0,1,0),d_flag);
  assign_maxfacet_edge<<<grid,block>>>(ext_rect,mk_cell(0,0,1),d_flag);

  assign_maxfacet_face<<<grid,block>>>(ext_rect,mk_cell(1,0,0),mk_cell(0,1,0),d_flag);
  assign_maxfacet_face<<<grid,block>>>(ext_rect,mk_cell(0,1,0),mk_cell(0,0,1),d_flag);
  assign_maxfacet_face<<<grid,block>>>(ext_rect,mk_cell(0,0,1),mk_cell(1,0,0),d_flag);

  assign_maxfacet_cube<<<grid,block>>>(ext_rect,d_flag);



  // pitched pointer
  hipPitchedPtr flag_pitched_ptr;
  flag_pitched_ptr.ptr      = flag;
  flag_pitched_ptr.pitch    = flag_vol_extent.width*sizeof(flag_t);
  flag_pitched_ptr.xsize    = flag_vol_extent.width;
  flag_pitched_ptr.ysize    = flag_vol_extent.height;

  // copy data to 3D array
  hipMemcpy3DParms flag_copy_params = {0};
  flag_copy_params.srcPtr   = flag_pitched_ptr;
  flag_copy_params.dstArray = d_flag;
  flag_copy_params.extent   = flag_vol_extent;
  flag_copy_params.kind     = hipMemcpyDeviceToHost;
  cutilSafeCall(hipMemcpy3D(&flag_copy_params));

  hipFreeArray(d_flag);
  hipFreeArray(d_func);
}

#define MAX(a,b) ((a > b) ? a : b)

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{ { 0x10,  8 },
	  { 0x11,  8 },
	  { 0x12,  8 },
	  { 0x13,  8 },
	  { 0x20, 32 },
	  { 0x21, 48 },
	  {   -1, -1 }
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}
// end of GPU Architecture definitions


// This function returns the best GPU (with maximum GFLOPS)
inline int cutGetMaxGflopsDeviceId()
{
	int current_device   = 0, sm_per_multiproc = 0;
	int max_compute_perf = 0, max_perf_device  = 0;
	int device_count     = 0, best_SM_arch     = 0;
	hipDeviceProp_t deviceProp;

	hipGetDeviceCount( &device_count );
	// Find the best major SM Architecture GPU device
	while ( current_device < device_count ) {
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major > 0 && deviceProp.major < 9999) {
			best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		}
		current_device++;
	}

    // Find the best CUDA capable GPU device
	current_device = 0;
	while( current_device < device_count ) {
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		    sm_per_multiproc = 1;
		} else {
			sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}

		int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
		if( compute_perf  > max_compute_perf ) {
            // If we find GPU with SM major > 2, search only these
			if ( best_SM_arch > 2 ) {
				// If our device==dest_SM_arch, choose this, or else pass
				if (deviceProp.major == best_SM_arch) {
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			} else {
				max_compute_perf  = compute_perf;
				max_perf_device   = current_device;
			}
		}
		++current_device;
	}
	return max_perf_device;
}

extern "C"
void init_cuda()
{
  hipSetDevice( cutGetMaxGflopsDeviceId() );
}


